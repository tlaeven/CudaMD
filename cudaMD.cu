
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

const int N = 108;
const int SIZE = N*3;
const float h = 0.004;
const float h2 = h/2;
const float L = pow(N/0.8, 1.0/3.0);
const int numBoxes = 2;
const int numBlocksAdd = SIZE;

void read_v(float b[SIZE])
{	
    FILE *fp = fopen("v_init","rb");
    size_t ret_code = fread(b, sizeof *b, SIZE, fp); // reads an array of floats
    if(ret_code == SIZE) {
        puts("Initial velocities read successfully");
        putchar('\n');
    } 
    else { // error handling
       if (feof(fp))
          printf("Error reading v_init: unexpected end of file\n");
       else if (ferror(fp)) {
          perror("Error reading v_init");
       }
    }
    fclose(fp);
}

void read_r(float b[SIZE])
{	
    FILE *fp = fopen("r_init","rb");
    size_t ret_code = fread(b, sizeof *b, SIZE, fp); // reads an array of floats
    if(ret_code == SIZE) {
        puts("Initial positions read successfully");
        putchar('\n');
    } 
    else { // error handling
       if (feof(fp))
          printf("Error reading r_init: unexpected end of file\n");
       else if (ferror(fp)) {
          perror("Error reading r_init");
       }
    }
    fclose(fp);
}

__global__ void vv_update_r(float F[SIZE], float r[SIZE], float v[SIZE])
{
    float L = 1.2;
	int i = blockIdx.x;
	r[i] = fmodf(r[i] + h2*F[i] + h*v[i],L);

}
__global__ void vv_update_v(float F[SIZE], float r[SIZE], float v[SIZE])
{
	int i = blockIdx.x;
	v[i] = v[i] + h2*F[i];
}

// RUN THIS FIRST TO CLEAR ALL PREVIOUS FORCES
__global__ void calcForces_intrabox(float F[SIZE], float r[SIZE], int boxMembers[N], int boxMembersFirstIndex[numBoxes+1], float L_tears[1])
{ 
  
  int block_A = blockIdx.x;
  int k = threadIdx.x; // every thread does multiple particles

  int N_par_thread = 4;
  float L = L_tears[0];
  
  int i = boxMembersFirstIndex[block_A];
  int N_A = boxMembersFirstIndex[block_A + 1] - i;
  
  ///////////////
  // SHARED MEMORY: PER BLOCK OF 32 THREADS(A WARP)
  ///////////////

  extern __shared__ float shared[];
  float *r_boxA = &shared[0];
  int counter = 3*N_A;
  float *F_boxA = &shared[counter];

  ///////////////
  // FILL TEMPORARY CONTAINER WITH PARTICLE POSITIONS
  ///////////////
  
  for (int t = N_par_thread*k; t < N_par_thread*(k+1); ++t)
    {
    if (t<N_A){  
      int l = 3*t; // particle number * 3 dimensions
      for (int n = 0; n < 3; ++n){
          r_boxA[l + n] = r[3*boxMembers[i+t] + n];
      }
    }
  }
__syncthreads(); // Make sure all boxes are filled
  
  ///////////////
  // FOUNTAIN OF TEARS
  ///////////////

  for (int t = N_par_thread*k; t < N_par_thread*(k+1); ++t)
  {

    if (t<N_A){
      int l = 3*t; // particle number * 3 dimensions
      // Fill artificial boxes with particle positions

      // Calc force
      float x_l = r_boxA[l];
      float y_l = r_boxA[l+1];
      float z_l = r_boxA[l+2];

      for (int n = 0; n < N_A; ++n)
      { 
        if (n==t)
        {
          continue;
        }
        int m = 3*n;

        float dx =  x_l - r_boxA[m];
        dx = dx - round(dx/L)*L;
        float dy =  y_l - r_boxA[m+1];
        dy = dy - round(dy/L)*L;
        float dz =  z_l - r_boxA[m+2];
        dz = dz - round(dz/L)*L;

        float R2 = dx*dx + dy*dy + dz*dz;

        float forceMagnitude = 48*pow(R2,-7) -24*pow(R2,-4);
        
        float fx = dx * forceMagnitude;
        F_boxA[l] += fx;
        
        float fy = dy * forceMagnitude;
        F_boxA[l+1] += fy;

        float fz = dz * forceMagnitude;
        F_boxA[l+2] += fz;

      }
    }
    }
  __syncthreads();

  ///////////////
  // REDISTRIBUTE FORCES INTO GLOBAL F
  ///////////////

  for(int t = N_par_thread*k; t < N_par_thread*(k+1); ++t){
      if (t<N_A)
      {
      for (int n = 0; n < 3; ++n){
        F[3*boxMembers[i+t] + n] = F_boxA[3*t + n];
      }
      }
    }
}
__global__ void update_Boxpair(float F[SIZE], float r[SIZE], int boxMembers[N], int boxMembersFirstIndex[numBoxes+1])
{
  int block_A = blockIdx.x;
  int block_B = blockIdx.x + 1; // horizontal example
  int t = threadIdx.x; // every thread does 1 particle in 

  float L = 5.129928;


  int l = 3*t; // particle number * 3 dimensions
  int i = boxMembersFirstIndex[block_A + 1];
  int j = boxMembersFirstIndex[block_B + 1];

  int N_A = boxMembersFirstIndex[block_A+1] - i;
  int N_B = boxMembersFirstIndex[block_B+1] - j;

  extern __shared__ float shared[];
  float *r_boxA = &shared[0];
  int counter = 3*N_A;
  float *r_boxB = &shared[counter];
  counter += 3*N_B;
  float *F_boxA = &shared[counter];
  counter += 3*N_A;
  float *F_boxB = &shared[counter];

   if (t<N_A){
    // Fill artificial boxes with particle positions
    for (int n = 0; n < 3; ++n){
        r_boxA[l + n] = r[boxMembers[i+t] + n];
        r_boxB[l + n] = r[boxMembers[j+t] + n];
    }
    __syncthreads(); // Make sure all boxes are filled


    // Calc force
    float x_l = r_boxA[l];
    float y_l = r_boxA[l+1];
    float z_l = r_boxA[l+2];
    for (int n = 0; n < N_B; ++n)
    { 
      int m = 3*n;

      float dx =  x_l - r_boxB[m];
      dx = dx - round(dx/L)*L;
      float dy =  y_l - r_boxB[m+1];
      dy = dy - round(dy/L)*L;
      float dz =  z_l - r_boxB[m+2];
      dz = dz - round(dz/L)*L;

      float R2 = dx*dx + dy*dy + dz*dz;


      float forceMagnitude = 48*pow(R2,-7) -24*pow(R2,-4);
      
      float fx = dx * forceMagnitude;
      F_boxA[l] += fx;
      F_boxB[m] -= fx;
      
      float fy = dy * forceMagnitude;
      F_boxA[l+1] += fy;
      F_boxB[m+1] -= fy;

      float fz = dz * forceMagnitude;
      F_boxA[l+2] += fz;
      F_boxB[m+2] -= fz;

    }


  __syncthreads(); // Make sure all forces have been filled

    for (int n = 0; n < 3; ++n){
      F[boxMembers[i+t] + n] += F_boxA[l + n];
      F[boxMembers[j+t] + n] += F_boxB[l + n];
  }
  }


}


void velocity_verlet(float F[SIZE], float r[SIZE], float v[SIZE])
{
	vv_update_r<<< numBlocksAdd, 1>>>(F, r, v);
  hipDeviceSynchronize();
	vv_update_v<<< numBlocksAdd, 1>>>(F, r, v);
	hipDeviceSynchronize();
  //calcForces
  // cudaDeviceSynchronize();
	vv_update_v<<< numBlocksAdd, 1>>>(F, r, v);
	hipDeviceSynchronize();
  }

int main(void)
{	
	float r0[SIZE], v0[SIZE], F0[SIZE], vout[SIZE];
	read_r(r0);
	read_v(v0);

  for (int i = 0; i < SIZE; ++i)
  {
    F0[i] = 1;
  }
  //read_v(F0); // fake F to test trivial vv kernels

  float* d_F0;
  hipMalloc(&d_F0, SIZE*sizeof(float));
  float* d_r0;
  hipMalloc(&d_r0, SIZE*sizeof(float));
  float* d_v0;
  hipMalloc(&d_v0, SIZE*sizeof(float));
  float * d_L;
  hipMalloc(&d_L, sizeof(float));
  
  hipMemcpy(d_F0, F0, SIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_r0, r0, SIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v0, v0, SIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_L, &L, sizeof(float), hipMemcpyHostToDevice);
  // for(int i=0; i<100;++i){
   // velocity_verlet(d_F0, d_r0, d_v0);
  // }

  int boxMembers[N];
  for (int i = 0; i < N; ++i)
  {
    boxMembers[i] = i;
  }
  int mbfi[numBoxes+1] = {0,54,N};

  int* d_boxMembers;
  hipMalloc(&d_boxMembers, N*sizeof(int));
  int* d_mbfi;
  hipMalloc(&d_mbfi, (numBoxes+1)*sizeof(int));
  hipMemcpy(d_boxMembers, boxMembers, N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_mbfi, mbfi, (numBoxes+1)*sizeof(int),hipMemcpyHostToDevice);

  // if random shit comes out, might be shared mem size
  for (int b = 0; b < 1000; ++b)
  {
    calcForces_intrabox<<<numBoxes, 28, SIZE*40>>>(d_F0, d_r0, d_boxMembers, d_mbfi, d_L);
  }
    //test_F<<<N,3>>>(d_F0);
  hipDeviceSynchronize();
	hipMemcpy(vout, d_F0, SIZE*sizeof(float), hipMemcpyDeviceToHost); // put in F0 to check if different to F0
	hipDeviceSynchronize();
  for(int i=0; i<N;++i){
		for(int j=0; j<3; ++j){
			printf("%f ",vout[3*i+j]);
		}
		printf("\t%i\n",i);
	}

}